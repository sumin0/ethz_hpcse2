#include "hip/hip_runtime.h"
#include "utils.h"
#include <algorithm>
#include <random>

__global__ void copyPermutedArray1(double *a, const double *b, const int *p){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 1024) a[idx] = b[p[idx]];
}

__global__ void copyPermutedArray2(double *a, const double *b, const int *p){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 1024) a[p[idx]] = b[idx];
}

__global__ void addVectors1(double *a, const double *b){ 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 1024) a[idx] = a[idx] + b[idx];
}

__global__ void addVectors2(double *a, const double *b){ 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i=0; i<100; i++){
        if (idx < 1024) a[idx] = a[idx] + b[idx];
    }
}

/// Buffer sizes we consider. The numbers are odd such that p[i]=(2*i)%K are all different.
static constexpr int kBufferSizes[] = {
    17, 65, 251, 1001, 2001, 5001,
    10'001, 25'001, 50'001, 100'001, 250'001, 500'001, 1'000'001,
    5'000'001, 20'000'001, 50'000'001,
};

void subtask_b() {
    constexpr int threadsPerBlock = 1024;
    int maxK = kBufferSizes[sizeof(kBufferSizes) / sizeof(kBufferSizes[0]) - 1];

    /// Pick a N with respect to K such that total running time is more or less uniform.
    auto pickN = [](int K) {
        return 100'000 / (int)std::sqrt(K) + 5;  // Some heuristics.
    };

    double *aDev;
    double *bDev;
    int *pDev;
    double *aHost;
    int *pHost;

    // TODO: Allocate the buffers. Immediately allocate large enough buffers to handle the largest case (maxK).
    // Wrap all cuda APIs with the CUDA_CHECK macro, which will report if the API failed to execute.
    // For example,
    //      CUDA_CHECK(hipMalloc(...));
    // CUDA_CHECK(cudaCmd) check whether `cudaCmd` completed successfully.
    CUDA_CHECK(hipMalloc(&aDev, maxK * sizeof(double)));
    CUDA_CHECK(hipMalloc(&bDev, maxK * sizeof(double)));
    CUDA_CHECK(hipMalloc(&pDev, maxK * sizeof(int)));
    CUDA_CHECK(hipHostMalloc(&aHost, maxK * sizeof(double)));
    CUDA_CHECK(hipHostMalloc(&pHost, maxK * sizeof(int)));
    //aHost = (double*)malloc(maxK * sizeof(double));
    //aHost = new double[maxK * sizeof(double)];
    //pHost = (int*)malloc(maxK * sizeof(int));
    //pHost = new int[maxK * sizeof(int)];

    // TODO: Delete this once done with allocation.
//   printf("Implement allocation first.\n");
//   return;


    // Set aDev, bDev and aHost to 0.0 (not really that important).
    CUDA_CHECK(hipMemset(aDev, 0, maxK * sizeof(double)));
    CUDA_CHECK(hipMemset(bDev, 0, maxK * sizeof(double)));
    memset(aHost, 0, maxK * sizeof(double));

    // Task 1b.1)
    for (int K : kBufferSizes) {
        // TODO: Measure the execution time of synchronously uploading K doubles from the host to the device. Report GB/s
        double dt = benchmark(pickN(K), [aDev, aHost, K](){
            CUDA_CHECK(hipMemcpy(aDev, aHost, K * sizeof(double), hipMemcpyHostToDevice));
        });
        double gbps = K * sizeof(double) * 1e-9 / dt; // Gigabytes per second here;
        printf("upload K=%8d --> %5.2f GB/s\n", K, gbps);
    }


    // Task 1b.2)
    /// Benchmark copying for a given access pattern (permutation).
    auto benchmarkPermutedCopy = [=](const char *description, auto permutationFunc) {
        for (int K : kBufferSizes) {
            // Compute the permutation p[i].
            permutationFunc(K);

            /// TODO: Copy pHost to pDev. Don't forget CUDA_CHECK.
            CUDA_CHECK(hipMemcpy(pDev, pHost, K * sizeof(int), hipMemcpyHostToDevice));
            /// TODO: Benchmark the a_i = b_{p_i} kernel.
            double dtABP = benchmark(pickN(K), [aDev, bDev, pDev, K](){
                CUDA_LAUNCH(copyPermutedArray1, (K + threadsPerBlock-1)/threadsPerBlock, threadsPerBlock, aDev, bDev, pDev);
                //CUDA_CHECK(hipDeviceSynchronize());
            });

            /// TODO: (OPTIONAL) Benchmark the a_{p_i} = b_i kernel;
            //double dtAPB = 0.0;
            double dtAPB = benchmark(pickN(K), [aDev, bDev, pDev, K](){
                CUDA_LAUNCH(copyPermutedArray2, (K + threadsPerBlock-1)/threadsPerBlock, threadsPerBlock, aDev, bDev, pDev);
	        //CUDA_CHECK(hipDeviceSynchronize());
            });

            // Report how many bytes per second was written.
            printf("Case %s  -->  K=%8d  [a=b_p] %6.2f GB/s  [a_p=b] %6.2f GB/s written\n",
                   description, K,
                   1e-9 * K * sizeof(double) / dtABP,
                   1e-9 * K * sizeof(double) / dtAPB);
        }
    };

    // The patterns are already implemented, do not modify!
    std::mt19937 gen;
    benchmarkPermutedCopy("p[i]=i", [pHost](int K) {
        for (int i = 0; i < K; ++i)
            pHost[i] = i;
    });
    benchmarkPermutedCopy("p[i]=(2*i)%K", [pHost](int K) {
        for (int i = 0; i < K; ++i)
            pHost[i] = (2 * i) % K;
    });
    benchmarkPermutedCopy("p[i]=(4*i)%K", [pHost](int K) {
        for (int i = 0; i < K; ++i)
            pHost[i] = (4 * i) % K;
    });
    benchmarkPermutedCopy("p[i]=i, 32-shuffled", [pHost, &gen](int K) {
        for (int i = 0; i < K; ++i)
            pHost[i] = i;
        for (int i = 0; i < K; i += 32)
            std::shuffle(pHost + i, pHost + std::min(i + 32, K), gen);
    });
    benchmarkPermutedCopy("fully shuffled", [pHost, &gen](int K) {
        for (int i = 0; i < K; ++i)
            pHost[i] = i;
        std::shuffle(pHost, pHost + K, gen);
    });


    // Task 1b.3) and 1b.4)
    for (int K : kBufferSizes) {
        // TODO: Benchmark a_i += b_i kernel.
        //double dt1 = 0.0;
        double dt1 = benchmark(pickN(K), [aDev, bDev, K](){
                CUDA_LAUNCH(addVectors1, (K + threadsPerBlock-1)/threadsPerBlock, threadsPerBlock, aDev, bDev);
        });
	// TODO: Benchmark the kernel that repeats a_i += b_i 100x times.
        double dt100 = benchmark(pickN(K), [aDev, bDev, K](){
                CUDA_LAUNCH(addVectors2, (K + threadsPerBlock-1)/threadsPerBlock, threadsPerBlock, aDev, bDev);
        });
        double gflops1 = K * 2 * 1e-9 / dt1;
        double gflops100 = 100 * K * 2 * 1e-9 / dt100;
        printf("a+b  1x -> %4.1f GFLOP/s  100x -> %5.1f GFLOP/s\n", gflops1, gflops100);
    }


    // TODO: Free all host and all device buffers.
    CUDA_CHECK(hipFree(aDev));
    CUDA_CHECK(hipFree(bDev));
    CUDA_CHECK(hipFree(pDev));
    CUDA_CHECK(hipHostFree(aHost));
    CUDA_CHECK(hipHostFree(pHost));
    //delete[] aHost;
    //delete[] pHost;
    //free(aHost);
    //free(pHost); 
}

int main() {
    subtask_b();
}
