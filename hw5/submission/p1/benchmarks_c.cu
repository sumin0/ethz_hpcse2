#include "hip/hip_runtime.h"
#include "utils.h"
#include <numeric>
#include <omp.h>
#include <vector>

using ll = long long;

// Compute the sum of the Leibniz series. Each thread takes care of a subset of terms.
__global__ void leibnizKernel(ll K, double *partialSums) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0.0;
    // TODO: Compute the partial sum. Pick however you like which terms are computed by which thread.
    //       Avoid using std::pow for computing (-1)^k!
    ll step = blockDim.x * gridDim.x;
    for(ll i=idx; i<K; i+=step){
        sum += (i % 2 == 0) ? 1.0 / (2.0*i+1.0) : (-1.0) / (2.0*i+1.0);
    }
    partialSums[idx] = sum;
}

/// Run the CUDA code for the given number of blocks and threads/block.
void runCUDA(ll K, int numBlocks, int threadsPerBlock) {
    int numThreads = numBlocks * threadsPerBlock;

    // Allocate the device and host buffers.

    double *partialSumsDev;
    double *partialSumsHost;

    // TODO: Allocate the temporary buffers for partial sums.
    CUDA_CHECK(hipMalloc(&partialSumsDev, numThreads * sizeof(double)));
    CUDA_CHECK(hipHostMalloc(&partialSumsHost, numThreads * sizeof(double)));

    // TODO: Run the kernel and benchmark execution time.
    double dt = benchmark(10, [=](){
           CUDA_LAUNCH(leibnizKernel, numBlocks, threadsPerBlock, K, partialSumsDev);
        });

    // TODO: Copy the sumsDev to host and accumulate, and sum them up.
    CUDA_CHECK(hipMemcpy(partialSumsHost, partialSumsDev, numThreads * sizeof(double), hipMemcpyDeviceToHost));
    double sum = std::accumulate(partialSumsHost, partialSumsHost + numThreads, 0.0);

    double pi = 4 * sum;
    printf("CUDA blocks=%5d  threads/block=%4d  iter/thread=%5lld  pi=%.12f  rel error=%.2g  Gterms/s=%.1f\n",
           numBlocks, threadsPerBlock, K / numThreads, pi, (pi - M_PI) / M_PI,
           1e-9 * K / dt);

    // TODO: Deallocate cuda buffers.
    CUDA_CHECK(hipFree(partialSumsDev));
    CUDA_CHECK(hipHostFree(partialSumsHost));
}

/// Run the OpenMP variant of the code.
void runOpenMP(ll K, int numThreads) {
    double sum = 0.0;
    double sign = 1.0;

    auto t0 = std::chrono::steady_clock::now();
    // TODO: Implement the Leibniz series summation with OpenMP.
#pragma omp parallel for reduction(+:sum) num_threads(numThreads)
    for(ll i=0; i<K; i++){
        sum += sign * 1 / (2*i+1);
        sign = sign * (-1);
    }

    // TODO: Benchmark execution time.
    auto t1 = std::chrono::steady_clock::now();
    double dt = 1e-9 * (double)std::chrono::duration_cast<std::chrono::nanoseconds>(t1-t0).count();

    double pi = 4 * sum;
    printf("OpenMP threads=%d  pi=%.16g  rel error=%.2g  Gterms/s=%.1f\n",
           numThreads, pi, (pi - M_PI) / M_PI, 1e-9 * K / dt);
};


void subtask_c() {
    constexpr ll K = 2LL << 30;

    // TODO: Experiment with number of threads per block, and number of blocks
    // (i.e. number of iterations per thread).
    runCUDA(K, 512, 512);

    runOpenMP(K, 12);
}

int main() {
    subtask_c();
}
