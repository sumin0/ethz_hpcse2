#include <hip/hip_runtime.h>

__global__ void computeForcesKernel(int N, const double3 *p, double3 *f) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    f[idx] = double3{0.0, 0.0, 0.0};
    for (int i = 0; i < N; ++i) {
        double dx = p[i].x - p[idx].x;
        double dy = p[i].y - p[idx].y;
        double dz = p[i].z - p[idx].z;
        // Instead of skipping the i == idx case, add 1e-150 to avoid division
        // by zero. (dx * inv_r will be exactly 0.0)
        double r = sqrt(1e-150 + dx * dx + dy * dy + dz * dz);
        double inv_r = 1 / r;
        f[idx].x += dx * inv_r * inv_r * inv_r;
        f[idx].y += dy * inv_r * inv_r * inv_r;
        f[idx].z += dz * inv_r * inv_r * inv_r;
    }
}

void computeForces(int N, const double3 *p, double3 *f) {
    constexpr int numThreads = 1024;
    int numBlocks = (N + numThreads - 1) / numThreads;
    computeForcesKernel<<<numBlocks, numThreads>>>(N, p, f);
}
