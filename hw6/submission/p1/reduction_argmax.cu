#include "hip/hip_runtime.h"
#include "utils.h"
#include <cassert>
#include <limits>

struct Pair {
    double max;
    int idx;
};

/// Find the maximum value `a` among all warps and return {max value, index of
/// the max}. The result must be correct on at least the 0th thread of each warp.
__device__ Pair argMaxWarp(double a) {
    // TODO: 1.b) Compute the argmax of the given value.
    //            Return the maximum and the location of the maximum (0..31).
    Pair result, new_result;

    unsigned int laneId = threadIdx.x & 0x1f;
    // unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    result.max = a;
    result.idx = laneId;
    
    for(unsigned int s = warpSize/2; s>0; s/=2){
        new_result.max = __shfl_down_sync(0xffffffff, result.max, s, warpSize);
        new_result.idx = __shfl_down_sync(0xffffffff, result.idx, s, warpSize);
        if (new_result.max > result.max){
            result.max = new_result.max;
            result.idx = new_result.idx;
        }
        else if ((new_result.max == result.max) && (new_result.idx < result.idx)){
            result.idx = new_result.idx;
        }
    }

    return result;
}


/// Returns the argmax of all values `a` within a block,
/// with the correct answer returned at least by the 0th thread of a block.
__device__ Pair argMaxBlock(double a) {
    // TODO: 1.c) Compute the argmax of the given value.
    //            Return the maximum and the location of the maximum (0..1023).
    // NOTE: For 1.c) implement either this or `sumBlock`!
    Pair result;
    result.max = 0.0;
    result.idx = 0;

    // ...

    return result;
}


void argMax1M(const double *aDev, Pair *bDev, int N) {
    assert(N <= 1024 * 1024);
    // TODO: 1.d) Implement either this or `sum1M`.
    //            Avoid copying any data back to the host.
    //            Hint: The solution requires more CUDA operations than just
    //            calling a single kernel. Feel free to use whatever you find
    //            necessary.
}

#include "reduction_argmax.h"

int main() {
    testSmallArgMax(argMaxWarpTestKernel, argMaxWarpCheck, 32, 3);
    testSmallArgMax(argMaxWarpTestKernel, argMaxWarpCheck, 32, 32);
    testSmallArgMax(argMaxWarpTestKernel, argMaxWarpCheck, 32, 320);
    testSmallArgMax(argMaxWarpTestKernel, argMaxWarpCheck, 32, 1023123);
    printf("argMaxWarp OK.\n");

    testSmallArgMax(argMaxBlockTestKernel, argMaxBlockCheck, 1024, 32);
    testSmallArgMax(argMaxBlockTestKernel, argMaxBlockCheck, 1024, 1024);
    testSmallArgMax(argMaxBlockTestKernel, argMaxBlockCheck, 1024, 12341);
    testSmallArgMax(argMaxBlockTestKernel, argMaxBlockCheck, 1024, 1012311);
    printf("argMaxBlock OK.\n");

    testLargeArgMax("argMax1M", argMax1M, 32);
    testLargeArgMax("argMax1M", argMax1M, 1024);
    testLargeArgMax("argMax1M", argMax1M, 12341);
    testLargeArgMax("argMax1M", argMax1M, 1012311);
    printf("argMax1M OK.\n");
}

